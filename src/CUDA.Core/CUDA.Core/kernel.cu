
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void changeTable(int *input, int *output, size_t pitch)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	output[i,j] = blockIdx.x;

}

int main()
{
	//allocation of matrix
	dim3 sizeOfDim(10,10);

	int *h_array;	
	int *d_array;
	int *output_array;

	size_t size = sizeOfDim.x * sizeOfDim.y * sizeof(int);

	size_t pitch;

	//allocation memory to host array
	h_array = (int*)malloc(size);

	for (int i = 0; i < sizeOfDim.x; i++)
	{
		for(int j = 0; j < sizeOfDim.y; j++)
		{
			h_array[i,j] = i+j;
		}
	}
	
	//allocation device array
	/*cudaMalloc(&d_array,size);
	cudaMalloc(&output_array, size);*/
	hipMallocPitch(&d_array,&pitch, sizeOfDim.x * sizeof(int), sizeOfDim.y);
	hipMallocPitch(&output_array, &pitch, sizeOfDim.x * sizeof(int), sizeOfDim.y);

		
	//copy data from host array to device array
	//cudaMemcpy(d_array, h_array, size, cudaMemcpyHostToDevice);
	hipMemcpy2D(d_array,pitch,h_array,sizeOfDim.x * sizeof(int),sizeOfDim.x * sizeof(int),sizeOfDim.y,hipMemcpyHostToDevice);


	//initialize block and threads
	dim3 threadsPerBlock(1,1);
	dim3 numberOfBlock(sizeOfDim.x / threadsPerBlock.x, sizeOfDim.y / threadsPerBlock.y);

	//do some cuda things
	changeTable<<<numberOfBlock,threadsPerBlock>>>(d_array, output_array, pitch);

	//copy result data from device to host
	//cudaMemcpy(h_array, output_array, size, cudaMemcpyDeviceToHost);
	hipMemcpy2D(h_array, sizeOfDim.x * sizeof(int), output_array, pitch, sizeOfDim.x * sizeof(int), sizeOfDim.y, hipMemcpyDeviceToHost);

	printf("Changed array \n");

	for (int i = 0; i < sizeOfDim.x; i++)
	{
		for (int j = 0; j < sizeOfDim.y; j++)
		{
			printf("%d \n",h_array[i,j]); 	

		}
		
	}	

	/*cudaFree(d_array);
	cudaFree(output_array);
	free(h_array);*/

	getchar();

}

